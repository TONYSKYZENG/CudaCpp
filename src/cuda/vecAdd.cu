//
// Created by tony on 16/12/22.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
__global__ void vecAdd_core(float *a, float *b, float *c, int n,int intensity)
{
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
  { float ru=0;
    float ta=0,tb=0;
    ta=a[id];
    tb=b[id];
    for(int k=0;k<intensity;k++)
    {
      ru+= ta+tb;
    }

    c[id] = ru;
  }

}
void vecAdd(float *h_a, float *h_b, float *h_c, int n, int intensity){
// Device input vectors
float *d_a;
float *d_b;
//Device output vector
float *d_c;
  // Size, in bytes, of each vector
  size_t bytes = n*sizeof(float);

  // Allocate memory for each vector on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  // Copy host vectors to device
  hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

  int blockSize, gridSize;

  // Number of threads in each thread block
  blockSize = 32;

  // Number of thread blocks in grid
  gridSize = (int)ceil((float)n/blockSize);

  // Execute the kernel
  vecAdd_core<<<gridSize, blockSize>>>(d_a, d_b, d_c, n,intensity);

  // Copy array back to host
  hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}